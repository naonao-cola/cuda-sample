﻿

/*
  This example demonstrates several CUTLASS utilities in the context of a mixed-precision
  floating-point matrix product computation.

  These utilities are intended to be useful supporting components for managing tensor and matrix
  memory allocations, initializing and comparing results, and computing reference output.

  CUTLASS utilities are defined in the directory `tools/util`, and definitions appear
  namespace `cutlass::` or an inner namespace therein. Operations in `cutlass::reference::` have
  both host-side and device-side implementations, and the choice to use device-side initialization
  and host-side verification in this example was arbitrary.

此示例在混合精度的上下文中演示了几个CUTLASS实用程序浮点矩阵乘积计算。

这些实用程序旨在成为管理张量和矩阵的有用支持组件内存分配、初始化和比较结果以及计算参考输出。

CUTLASS实用程序在目录“tools/util”中定义，定义出现在命名空间`cutlass:：`或其中的内部命名空间。在`cutlass:：reference:：`中的操作有
主机端和设备端实现，以及使用设备端初始化的选择并且该示例中的主机侧验证是任意的。


  cutlass::half_t

    This is a numeric type implementing IEEE half-precision quantities. It is functional in host
    and device code. In host-side code, CUTLASS_ENABLE_F16C optionally enables harware-accelerated
    numeric conversion on x86-64 CPUs support F16C extensions. In device code, all available
    hardware is used to implement conversion and numeric operations.

这是一种实现IEEE半精度量的数字类型。它在主机中正常工作以及设备代码。在主机端代码中，CUTALSS_ENABLE_F16C可选地启用硬件加速
x86-64 CPU上的数字转换支持F16C扩展。在设备代码中，所有可用硬件用于实现转换和数值运算。

  cutlass::HostTensor<>

    This template class simplifies the creation of tensors for all supported layouts. It simplifies
    allocation and management of host- and device- memory allocations.

    This class offers methods device_view() and host_view() to provide TensorView objects for
    device- and host-side memory allocations.

此模板类简化了为所有支持的布局创建张量的过程。它简化了主机和设备内存分配的分配和管理。
此类提供了device_view()和host_view()设备和主机侧内存分配。

  cutlass::reference::device::TensorFillRandomGaussian()

    This template function initializes elementsof a tensor to a random Gaussian distribution. It
    uses cuRAND in device code to compute random numbers.

此模板函数将张量的元素初始化为随机高斯分布。它在设备代码中使用cuRAND来计算随机数。

  cutlass::reference::host::Gemm<>

    This template function computes the general matrix product. This template supports unique
    data types for each matrix operand, the internal accumulation type, and the scalar parameters
    alpha and beta.


  cutlass::reference::host::TensorEquals()

    Compares two tensors of identical rank and returns true if values are bit equivalent.

*/

// Standard Library includes
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>

// CUTLASS includes needed for half-precision GEMM kernel
#include "cutlass/core_io.h"
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/layout/matrix.h"

//
// CUTLASS utility includes
//

// Defines operator<<() to write TensorView objects to std::ostream
#include "cutlass/util/tensor_view_io.h"

// Defines cutlass::HostTensor<>
#include "cutlass/util/host_tensor.h"

// Defines cutlass::half_t
#include "cutlass/numeric_types.h"

// Defines device_memory::copy_device_to_device()
#include "cutlass/util/device_memory.h"

// Defines cutlass::reference::device::TensorFillRandomGaussian()
#include "cutlass/util/reference/device/tensor_fill.h"

// Defines cutlass::reference::host::TensorEquals()
#include "cutlass/util/reference/host/tensor_compare.h"

// Defines cutlass::reference::host::Gemm()
#include "cutlass/util/reference/host/gemm.h"

#pragma warning(disable : 4503)
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t cutlass_hgemm_nn(int M, int N, int K, cutlass::half_t alpha, cutlass::half_t const* A, cutlass::layout::ColumnMajor::Stride::Index lda, cutlass::half_t const* B,
                             cutlass::layout::ColumnMajor::Stride::Index ldb, cutlass::half_t beta, cutlass::half_t* C, cutlass::layout::ColumnMajor::Stride::Index ldc)
{

    // Define the GEMM operation
    using Gemm = cutlass::gemm::device::Gemm<cutlass::half_t,                // ElementA
                                             cutlass::layout::ColumnMajor,   // LayoutA
                                             cutlass::half_t,                // ElementB
                                             cutlass::layout::ColumnMajor,   // LayoutB
                                             cutlass::half_t,                // ElementOutput
                                             cutlass::layout::ColumnMajor    // LayoutOutput
                                             >;

    Gemm gemm_op;

    cutlass::Status status = gemm_op({{M, N, K}, {A, lda}, {B, ldb}, {C, ldc}, {C, ldc}, {alpha, beta}});

    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
hipError_t TestCutlassGemm(int M, int N, int K, cutlass::half_t alpha, cutlass::half_t beta)
{
    hipError_t result;

    //
    // Construct cutlass::HostTensor<> using the half-precision host-side type.
    //
    // cutlass::HostTensor<> allocates memory on both the host and device corresponding to rank=2
    // tensors in column-major layout. Explicit synchronization methods are offered to copy the
    // tensor to the device or to the host.
    //

    // M-by-K matrix of cutlass::half_t
    cutlass::HostTensor<cutlass::half_t, cutlass::layout::ColumnMajor> A(cutlass::MatrixCoord(M, K));

    // K-by-N matrix of cutlass::half_t
    cutlass::HostTensor<cutlass::half_t, cutlass::layout::ColumnMajor> B(cutlass::MatrixCoord(K, N));

    // M-by-N matrix of cutlass::half_t
    cutlass::HostTensor<cutlass::half_t, cutlass::layout::ColumnMajor> C_cutlass(cutlass::MatrixCoord(M, N));

    // M-by-N matrix of cutlass::half_t
    cutlass::HostTensor<cutlass::half_t, cutlass::layout::ColumnMajor> C_reference(cutlass::MatrixCoord(M, N));

    //
    // Initialize matrices with small, random integers.
    //

    // Arbitrary RNG seed value. Hard-coded for deterministic results.
    uint64_t seed = 2080;

    // Gaussian random distribution
    cutlass::half_t mean   = 0.0_hf;
    cutlass::half_t stddev = 5.0_hf;

    // Specify the number of bits right of the binary decimal that are permitted
    // to be non-zero. A value of "0" here truncates random values to integers
    int bits_less_than_one = 0;

    cutlass::reference::device::TensorFillRandomGaussian(A.device_view(), seed, mean, stddev, bits_less_than_one);

    cutlass::reference::device::TensorFillRandomGaussian(B.device_view(), seed * 2019, mean, stddev, bits_less_than_one);

    cutlass::reference::device::TensorFillRandomGaussian(C_cutlass.device_view(), seed * 1993, mean, stddev, bits_less_than_one);


    // Copy C_cutlass into C_reference so the GEMM is correct when beta != 0.
    cutlass::device_memory::copy_device_to_device(C_reference.device_data(), C_cutlass.device_data(), C_cutlass.capacity());

    // Copy the device-side view into host memory
    C_reference.sync_host();

    //
    // Launch the CUTLASS GEMM kernel
    //

    result = cutlass_hgemm_nn(M, N, K, alpha, A.device_data(), A.stride(0), B.device_data(), B.stride(0), beta, C_cutlass.device_data(), C_cutlass.stride(0));

    if (result != hipSuccess) {
        return result;
    }

    //
    // Verify the result using a host-side reference
    //

    // A and B were initialized using device-side procedures. The intent of this example is to
    // use the host-side reference GEMM, so we must perform a device-to-host copy.
    A.sync_host();
    B.sync_host();

    // Copy CUTLASS's GEMM results into host memory.
    C_cutlass.sync_host();

    // Compute the reference result using the host-side GEMM reference implementation.
    cutlass::reference::host::Gemm<cutlass::half_t,                // ElementA
                                   cutlass::layout::ColumnMajor,   // LayoutA
                                   cutlass::half_t,                // ElementB
                                   cutlass::layout::ColumnMajor,   // LayoutB
                                   cutlass::half_t,                // ElementOutput
                                   cutlass::layout::ColumnMajor,   // LayoutOutput
                                   cutlass::half_t,
                                   cutlass::half_t>
        gemm_ref;

    gemm_ref({M, N, K},               // problem size (type: cutlass::gemm::GemmCoord)
             alpha,                   // alpha        (type: cutlass::half_t)
             A.host_ref(),            // A            (type: TensorRef<half_t, ColumnMajor>)
             B.host_ref(),            // B            (type: TensorRef<half_t, ColumnMajor>)
             beta,                    // beta         (type: cutlass::half_t)
             C_reference.host_ref()   // C            (type: TensorRef<half_t, ColumnMajor>)
    );

    // Compare reference to computed results.
    if (!cutlass::reference::host::TensorEquals(C_reference.host_view(), C_cutlass.host_view())) {

        char const* filename = "errors_01_cutlass_utilities.csv";

        std::cerr << "Error - CUTLASS GEMM kernel differs from reference. Wrote computed and reference results to '" << filename << "'" << std::endl;

        //
        // On error, print C_cutlass and C_reference to std::cerr.
        //
        // Note, these are matrices of half-precision elements stored in host memory as
        // arrays of type cutlass::half_t.
        //

        std::ofstream file(filename);

        // Result of CUTLASS GEMM kernel
        file << "\n\nCUTLASS =\n" << C_cutlass.host_view() << std::endl;

        // Result of reference computation
        file << "\n\nReference =\n" << C_reference.host_view() << std::endl;

        // Return error code.
        return hipErrorUnknown;
    }

    // Passed error check
    return hipSuccess;
}


/// Entry point to cutlass_utilities example.
//
// usage:
//
//   01_cutlass_utilities <M> <N> <K> <alpha> <beta>
//
int main(int argc, const char* arg[])
{

    //
    // This example uses half-precision and is only suitable for devices with compute capabitliy 5.3 or greater.
    //

    hipDeviceProp_t prop;
    hipError_t    result = hipGetDeviceProperties(&prop, 0);

    if (result != hipSuccess) {
        std::cerr << "Failed to query device properties with error " << hipGetErrorString(result) << std::endl;
        return -1;
    }

    if (!(prop.major > 5 || (prop.major == 5 && prop.minor >= 3))) {
        std::cerr << "This example uses half precision and is only suitable for devices with compute capability 5.3 or greater.\n";
        std::cerr << "You are using a CUDA device with compute capability " << prop.major << "." << prop.minor << std::endl;
        return -1;
    }

    //
    // Parse the command line to obtain GEMM dimensions and scalar values.
    //

    // GEMM problem dimensions: <M> <N> <K>
    int problem[3] = {128, 128, 128};

    for (int i = 1; i < argc && i < 4; ++i) {
        std::stringstream ss(arg[i]);
        ss >> problem[i - 1];
    }

    // Linear scale factors in GEMM. Note, these are half-precision values stored as
    // cutlass::half_t.
    //
    // Values outside the range of IEEE FP16 will overflow to infinity or underflow to zero.
    //
    cutlass::half_t scalars[2] = {1.0_hf, 0.0_hf};

    for (int i = 4; i < argc && i < 6; ++i) {
        std::stringstream ss(arg[i]);

        ss >> scalars[i - 4];   // lexical cast to cutlass::half_t
    }

    //
    // Run the CUTLASS GEMM test.
    //

    result = TestCutlassGemm(problem[0],   // GEMM M dimension
                             problem[1],   // GEMM N dimension
                             problem[2],   // GEMM K dimension
                             scalars[0],   // alpha
                             scalars[1]    // beta
    );

    if (result == hipSuccess) {
        std::cout << "Passed." << std::endl;
    }

    // Exit.
    return result == hipSuccess ? 0 : -1;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
