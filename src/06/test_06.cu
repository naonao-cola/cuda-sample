#include "hip/hip_runtime.h"
﻿#include "test_06.cuh"


__global__ void kernel(float* g_data, float value)
{
    int idx     = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + value;
}

int checkResult(float* data, const int n, const float x)
{
    for (int i = 0; i < n; i++) {
        if (data[i] != x) {
            printf("Error! data[%d] = %f, ref = %f\n", i, data[i], x);
            return 0;
        }
    }

    return 1;
}

void asyncAPI()
{
    // int            devID = 0;
    // hipDeviceProp_t deviceProps;
    // CHECK(hipGetDeviceProperties(&deviceProps, devID));
    // printf("> %s running on", argv[0]);
    // printf(" CUDA device [%s]\n", deviceProps.name);

    int   num    = 1 << 24;
    int   nbytes = num * sizeof(int);
    float value  = 10.0f;

    // allocate host memory
    float* h_a = 0;
    CHECK(hipHostMalloc((void**)&h_a, nbytes));
    memset(h_a, 0, nbytes);

    // allocate device memory
    float* d_a = 0;
    CHECK(hipMalloc((void**)&d_a, nbytes));
    CHECK(hipMemset(d_a, 255, nbytes));

    // set kernel launch configuration
    dim3 block = dim3(512);
    dim3 grid  = dim3((num + block.x - 1) / block.x);

    // create cuda event handles
    hipEvent_t stop;
    CHECK(hipEventCreate(&stop));

    // asynchronously issue work to the GPU (all to stream 0)
    CHECK(hipMemcpyAsync(d_a, h_a, nbytes, hipMemcpyHostToDevice));
    kernel<<<grid, block>>>(d_a, value);
    CHECK(hipMemcpyAsync(h_a, d_a, nbytes, hipMemcpyDeviceToHost));
    CHECK(hipEventRecord(stop));

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter = 0;

    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }

    // print the cpu and gpu times
    printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter);

    // check the output for correctness
    bool bFinalResults = (bool)checkResult(h_a, num, value);

    // release resources
    CHECK(hipEventDestroy(stop));
    CHECK(hipHostFree(h_a));
    CHECK(hipFree(d_a));
    CHECK(hipDeviceReset());
    // exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}

#define N 300000
#define NSTREAM 4

void CUDART_CB my_callback(hipStream_t stream, hipError_t status, void* data)
{
    printf("callback from stream %d\n", *((int*)data));
}

__global__ void kernel_1()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_2()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_3()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_4()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

void simpleCallback()
{
    int n_streams = NSTREAM;

    int            dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    // printf("> %s Starting...\n", argv[0]);
    printf("> Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5)) {
        if (deviceProp.concurrentKernels == 0) {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                   "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // set up max connectioin
    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv(iname, "8", 1);
    char* ivalue = getenv(iname);
    printf("> %s = %s\n", iname, ivalue);
    printf("> with streams = %d\n", n_streams);

    // Allocate and initialize an array of stream handles
    hipStream_t* streams = (hipStream_t*)malloc(n_streams * sizeof(hipStream_t));

    for (int i = 0; i < n_streams; i++) {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    dim3        block(1);
    dim3        grid(1);
    hipEvent_t start_event, stop_event;
    CHECK(hipEventCreate(&start_event));
    CHECK(hipEventCreate(&stop_event));

    int stream_ids[n_streams];

    CHECK(hipEventRecord(start_event, 0));

    for (int i = 0; i < n_streams; i++) {
        stream_ids[i] = i;
        kernel_1<<<grid, block, 0, streams[i]>>>();
        kernel_2<<<grid, block, 0, streams[i]>>>();
        kernel_3<<<grid, block, 0, streams[i]>>>();
        kernel_4<<<grid, block, 0, streams[i]>>>();
        CHECK(hipStreamAddCallback(streams[i], my_callback, (void*)(stream_ids + i), 0));
    }
    //事件同步函数
    CHECK(hipEventRecord(stop_event, 0));
    CHECK(hipEventSynchronize(stop_event));

    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start_event, stop_event));
    printf("Measured time for parallel execution = %.3fs\n", elapsed_time / 1000.0f);

    // release all stream
    for (int i = 0; i < n_streams; i++) {
        CHECK(hipStreamDestroy(streams[i]));
    }

    free(streams);

    /*
     * hipDeviceReset must be called before exiting in order for profiling and
     * tracing tools such as Nsight and Visual Profiler to show complete traces.
     */
    CHECK(hipDeviceReset());
}

void simpleHyperqBreadth(int argv1, int argv2)
{
    int n_streams = NSTREAM;
    int isize     = 1;
    int iblock    = 1;
    int bigcase   = 1;

    // get argument from command line
    if (argv1 > 1)
        n_streams = argv1;

    if (argv2 > 1)
        bigcase = argv2;

    float elapsed_time;

    // set up max connectioin
    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv(iname, "32", 1);
    char* ivalue = getenv(iname);
    printf("%s = %s\n", iname, ivalue);

    int            dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s with num_streams %d\n", dev, deviceProp.name, n_streams);
    CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5)) {
        if (deviceProp.concurrentKernels == 0) {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // Allocate and initialize an array of stream handles
    hipStream_t* streams = (hipStream_t*)malloc(n_streams * sizeof(hipStream_t));

    for (int i = 0; i < n_streams; i++) {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    // run kernel with more threads
    if (bigcase == 1) {
        iblock = 512;
        isize  = 1 << 12;
    }

    // set up execution configuration
    dim3 block(iblock);
    dim3 grid(isize / iblock);
    printf("> grid %d block %d\n", grid.x, block.x);

    // creat events
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // record start event
    CHECK(hipEventRecord(start, 0));

    // dispatch job with breadth first ordering
    //以广度优先排序调度作业
    for (int i = 0; i < n_streams; i++)
        kernel_1<<<grid, block, 0, streams[i]>>>();

    for (int i = 0; i < n_streams; i++)
        kernel_2<<<grid, block, 0, streams[i]>>>();

    for (int i = 0; i < n_streams; i++)
        kernel_3<<<grid, block, 0, streams[i]>>>();

    for (int i = 0; i < n_streams; i++)
        kernel_4<<<grid, block, 0, streams[i]>>>();

    // record stop event
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution = %.3fs\n", elapsed_time / 1000.0f);

    // release all stream
    for (int i = 0; i < n_streams; i++) {
        CHECK(hipStreamDestroy(streams[i]));
    }

    free(streams);

    // destroy events
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    // reset device
    CHECK(hipDeviceReset());
}

void simpleHyperqDependence(int argv1, int argv2)
{
    int n_streams = NSTREAM;
    int isize     = 1;
    int iblock    = 1;
    int bigcase   = 1;

    // get argument from command line
    if (argv1 > 1)
        n_streams = argv1;

    if (argv1 > 1)
        bigcase = argv2;

    float elapsed_time;

    // set up max connectioin
    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv(iname, "32", 1);
    char* ivalue = getenv(iname);
    printf("%s = %s\n", iname, ivalue);

    int            dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s with num_streams %d\n", dev, deviceProp.name, n_streams);
    CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5)) {
        if (deviceProp.concurrentKernels == 0) {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                   "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // Allocate and initialize an array of stream handles
    hipStream_t* streams = (hipStream_t*)malloc(n_streams * sizeof(hipStream_t));

    for (int i = 0; i < n_streams; i++) {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    // run kernel with more threads
    if (bigcase == 1) {
        iblock = 512;
        isize  = 1 << 12;
    }

    // set up execution configuration
    dim3 block(iblock);
    dim3 grid(isize / iblock);
    printf("> grid %d block %d\n", grid.x, block.x);

    // creat events
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));


    hipEvent_t* kernelEvent;
    kernelEvent = (hipEvent_t*)malloc(n_streams * sizeof(hipEvent_t));

    for (int i = 0; i < n_streams; i++) {
        CHECK(hipEventCreateWithFlags(&(kernelEvent[i]), hipEventDisableTiming));
    }

    // record start event
    CHECK(hipEventRecord(start, 0));

    // dispatch job with depth first ordering
    for (int i = 0; i < n_streams; i++) {
        kernel_1<<<grid, block, 0, streams[i]>>>();
        kernel_2<<<grid, block, 0, streams[i]>>>();
        kernel_3<<<grid, block, 0, streams[i]>>>();
        kernel_4<<<grid, block, 0, streams[i]>>>();

        CHECK(hipEventRecord(kernelEvent[i], streams[i]));
        //这个函数会阻塞设置的流，直到指定的事件在设备上发生。一旦事件发生，流中的后续操作可以继续执行。
        CHECK(hipStreamWaitEvent(streams[n_streams - 1], kernelEvent[i], 0));
    }

    // record stop event
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution = %.3fs\n", elapsed_time / 1000.0f);

    // release all stream
    for (int i = 0; i < n_streams; i++) {
        CHECK(hipStreamDestroy(streams[i]));
        CHECK(hipEventDestroy(kernelEvent[i]));
    }

    free(streams);
    free(kernelEvent);

    // reset device
    CHECK(hipDeviceReset());
}

void simpleHyperqDepth(int argv1, int argv2)
{
    int n_streams = NSTREAM;
    int isize     = 1;
    int iblock    = 1;
    int bigcase   = 1;

    // get argument from command line
    if (argv1 > 1)
        n_streams = argv1;

    if (argv2 > 1)
        bigcase = argv2;

    float elapsed_time;

    // set up max connectioin
    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv(iname, "32", 1);
    char* ivalue = getenv(iname);
    printf("%s = %s\n", iname, ivalue);

    int            dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s with num_streams=%d\n", dev, deviceProp.name, n_streams);
    CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5)) {
        if (deviceProp.concurrentKernels == 0) {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                   "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // Allocate and initialize an array of stream handles
    hipStream_t* streams = (hipStream_t*)malloc(n_streams * sizeof(hipStream_t));

    for (int i = 0; i < n_streams; i++) {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    // run kernel with more threads
    if (bigcase == 1) {
        iblock = 512;
        isize  = 1 << 12;
    }

    // set up execution configuration
    dim3 block(iblock);
    dim3 grid(isize / iblock);
    printf("> grid %d block %d\n", grid.x, block.x);

    // creat events
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // record start event
    CHECK(hipEventRecord(start, 0));

    // dispatch job with depth first ordering
    for (int i = 0; i < n_streams; i++) {
        kernel_1<<<grid, block, 0, streams[i]>>>();
        kernel_2<<<grid, block, 0, streams[i]>>>();
        kernel_3<<<grid, block, 0, streams[i]>>>();
        kernel_4<<<grid, block, 0, streams[i]>>>();
    }

    // record stop event
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution = %.3fs\n", elapsed_time / 1000.0f);

    // release all stream
    for (int i = 0; i < n_streams; i++) {
        CHECK(hipStreamDestroy(streams[i]));
    }

    free(streams);

    // destroy events
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    // reset device
    CHECK(hipDeviceReset());
}

#include "omp.h"
void simpleHyperqOpenmp()
{
    int n_streams = NSTREAM;
    int isize     = 1;
    int iblock    = 1;
    int bigcase   = 1;

    // get argument from command line
    // if (argc > 1)
    //     n_streams = atoi(argv[1]);

    // if (argc > 2)
    //     bigcase = atoi(argv[2]);

    float elapsed_time;

    // set up max connectioin
    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv(iname, "32", 1);
    char* ivalue = getenv(iname);
    printf("%s = %s\n", iname, ivalue);

    int            dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s with num_streams=%d\n", dev, deviceProp.name, n_streams);
    CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5)) {
        if (deviceProp.concurrentKernels == 0) {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                   "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // Allocate and initialize an array of stream handles
    hipStream_t* streams = (hipStream_t*)malloc(n_streams * sizeof(hipStream_t));

    for (int i = 0; i < n_streams; i++) {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    // run kernel with more threads
    if (bigcase == 1) {
        iblock = 512;
        isize  = 1 << 12;
    }

    // set up execution configuration
    dim3 block(iblock);
    dim3 grid(isize / iblock);
    printf("> grid %d block %d\n", grid.x, block.x);

    // creat events
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // record start event
    CHECK(hipEventRecord(start, 0));

    // dispatch job with depth first ordering using OpenMP
    omp_set_num_threads(n_streams);
#pragma omp parallel
    {
        int i = omp_get_thread_num();
        kernel_1<<<grid, block, 0, streams[i]>>>();
        kernel_2<<<grid, block, 0, streams[i]>>>();
        kernel_3<<<grid, block, 0, streams[i]>>>();
        kernel_4<<<grid, block, 0, streams[i]>>>();
    }

    // record stop event
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution = %.3fs\n", elapsed_time / 1000.0f);

    // release all stream
    for (int i = 0; i < n_streams; i++) {
        CHECK(hipStreamDestroy(streams[i]));
    }

    free(streams);

    // destroy events
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    // reset device
    CHECK(hipDeviceReset());
}

#define NSTREAM 4
#define BDIM 128

void initialData(float* ip, int size)
{
    int i;

    for (i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumArraysOnHost(float* A, float* B, float* C, const int n)
{
    for (int idx = 0; idx < n; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArrays(float* A, float* B, float* C, const int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        for (int i = 0; i < n; ++i) {
            C[idx] = A[idx] + B[idx];
        }
    }
}

void checkResult(float* hostRef, float* gpuRef, const int n)
{
    double epsilon = 1.0E-8;
    bool   match   = 1;

    for (int i = 0; i < n; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
}


void simpleMultiAddBreadth()
{
    int            dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5)) {
        if (deviceProp.concurrentKernels == 0) {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                   "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // set up max connectioin
    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv(iname, "1", 1);
    char* ivalue = getenv(iname);
    printf("> %s = %s\n", iname, ivalue);
    printf("> with streams = %d\n", NSTREAM);

    // set up data size of vectors
    int nElem = 1 << 18;
    printf("> vector size = %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // malloc pinned host memory for async memcpy
    float *h_A, *h_B, *hostRef, *gpuRef;
    CHECK(hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&h_B, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&hostRef, nBytes, hipHostMallocDefault));

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // invoke kernel at host side
    dim3 block(BDIM);
    dim3 grid((nElem + block.x - 1) / block.x);
    printf("> grid (%d, %d) block (%d, %d)\n", grid.x, grid.y, block.x, block.y);

    // sequential operation
    //顺序操作
    CHECK(hipEventRecord(start, 0));
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float memcpy_h2d_time;
    CHECK(hipEventElapsedTime(&memcpy_h2d_time, start, stop));

    CHECK(hipEventRecord(start, 0));
    sumArrays<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float kernel_time;
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipEventRecord(start, 0));
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float memcpy_d2h_time;
    CHECK(hipEventElapsedTime(&memcpy_d2h_time, start, stop));
    float itotal = kernel_time + memcpy_h2d_time + memcpy_d2h_time;

    printf("\n");
    printf("Measured timings (throughput):\n");
    printf(" Memcpy host to device\t: %f ms (%f GB/s)\n", memcpy_h2d_time, (nBytes * 1e-6) / memcpy_h2d_time);
    printf(" Memcpy device to host\t: %f ms (%f GB/s)\n", memcpy_d2h_time, (nBytes * 1e-6) / memcpy_d2h_time);
    printf(" Kernel\t\t\t: %f ms (%f GB/s)\n", kernel_time, (nBytes * 2e-6) / kernel_time);
    printf(" Total\t\t\t: %f ms (%f GB/s)\n", itotal, (nBytes * 2e-6) / itotal);

    // grid parallel operation
    int    iElem  = nElem / NSTREAM;
    size_t iBytes = iElem * sizeof(float);
    grid.x        = (iElem + block.x - 1) / block.x;

    hipStream_t stream[NSTREAM];

    for (int i = 0; i < NSTREAM; ++i) {
        CHECK(hipStreamCreate(&stream[i]));
    }

    CHECK(hipEventRecord(start, 0));

    // initiate all asynchronous transfers to the device
    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem;
        CHECK(hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]));
        CHECK(hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]));
    }

    // launch a kernel in each stream
    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem;
        sumArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset], &d_C[ioffset], iElem);
    }

    // enqueue asynchronous transfers from the device
    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem;
        CHECK(hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes, hipMemcpyDeviceToHost, stream[i]));
    }

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float execution_time;
    CHECK(hipEventElapsedTime(&execution_time, start, stop));

    printf("\n");
    printf("Actual results from overlapped data transfers:\n");
    printf(" overlap with %d streams : %f ms (%f GB/s)\n", NSTREAM, execution_time, (nBytes * 2e-6) / execution_time);
    printf(" speedup                : %f \n", ((itotal - execution_time) * 100.0f) / itotal);

    // check kernel error
    CHECK(hipGetLastError());

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    CHECK(hipHostFree(h_A));
    CHECK(hipHostFree(h_B));
    CHECK(hipHostFree(hostRef));
    CHECK(hipHostFree(gpuRef));

    // destroy events
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    // destroy streams
    for (int i = 0; i < NSTREAM; ++i) {
        CHECK(hipStreamDestroy(stream[i]));
    }

    CHECK(hipDeviceReset());
}

void simpleMultiAddDepth()
{
    int            dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5)) {
        if (deviceProp.concurrentKernels == 0) {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                   "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // set up max connectioin
    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv(iname, "1", 1);
    char* ivalue = getenv(iname);
    printf("> %s = %s\n", iname, ivalue);
    printf("> with streams = %d\n", NSTREAM);

    // set up data size of vectors
    int nElem = 1 << 18;
    printf("> vector size = %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // malloc pinned host memory for async memcpy
    float *h_A, *h_B, *hostRef, *gpuRef;
    CHECK(hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&h_B, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&hostRef, nBytes, hipHostMallocDefault));

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // invoke kernel at host side
    dim3 block(BDIM);
    dim3 grid((nElem + block.x - 1) / block.x);
    printf("> grid (%d, %d) block (%d, %d)\n", grid.x, grid.y, block.x, block.y);

    // sequential operation
    CHECK(hipEventRecord(start, 0));
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float memcpy_h2d_time;
    CHECK(hipEventElapsedTime(&memcpy_h2d_time, start, stop));

    CHECK(hipEventRecord(start, 0));
    sumArrays<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float kernel_time;
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipEventRecord(start, 0));
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float memcpy_d2h_time;
    CHECK(hipEventElapsedTime(&memcpy_d2h_time, start, stop));
    float itotal = kernel_time + memcpy_h2d_time + memcpy_d2h_time;

    printf("\n");
    printf("Measured timings (throughput):\n");
    printf(" Memcpy host to device\t: %f ms (%f GB/s)\n", memcpy_h2d_time, (nBytes * 1e-6) / memcpy_h2d_time);
    printf(" Memcpy device to host\t: %f ms (%f GB/s)\n", memcpy_d2h_time, (nBytes * 1e-6) / memcpy_d2h_time);
    printf(" Kernel\t\t\t: %f ms (%f GB/s)\n", kernel_time, (nBytes * 2e-6) / kernel_time);
    printf(" Total\t\t\t: %f ms (%f GB/s)\n", itotal, (nBytes * 2e-6) / itotal);

    // grid parallel operation
    int    iElem  = nElem / NSTREAM;
    size_t iBytes = iElem * sizeof(float);
    grid.x        = (iElem + block.x - 1) / block.x;

    hipStream_t stream[NSTREAM];

    for (int i = 0; i < NSTREAM; ++i) {
        CHECK(hipStreamCreate(&stream[i]));
    }

    CHECK(hipEventRecord(start, 0));

    // initiate all work on the device asynchronously in depth-first order
    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem;
        CHECK(hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]));
        CHECK(hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]));
        sumArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset], &d_C[ioffset], iElem);
        CHECK(hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes, hipMemcpyDeviceToHost, stream[i]));
    }

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float execution_time;
    CHECK(hipEventElapsedTime(&execution_time, start, stop));

    printf("\n");
    printf("Actual results from overlapped data transfers:\n");
    printf(" overlap with %d streams : %f ms (%f GB/s)\n", NSTREAM, execution_time, (nBytes * 2e-6) / execution_time);
    printf(" speedup                : %f \n", ((itotal - execution_time) * 100.0f) / itotal);

    // check kernel error
    CHECK(hipGetLastError());

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    CHECK(hipHostFree(h_A));
    CHECK(hipHostFree(h_B));
    CHECK(hipHostFree(hostRef));
    CHECK(hipHostFree(gpuRef));

    // destroy events
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    // destroy streams
    for (int i = 0; i < NSTREAM; ++i) {
        CHECK(hipStreamDestroy(stream[i]));
    }

    CHECK(hipDeviceReset());
}