#include "hip/hip_runtime.h"
﻿#include "test_03.cuh"

void simpleDeviceQuery()
{
    int            iDev = 0;
    hipDeviceProp_t iProp;
    CHECK(hipGetDeviceProperties(&iProp, iDev));
    printf("Device %d: %s\n", iDev, iProp.name);
    printf("  Number of multiprocessors:                     %d\n", iProp.multiProcessorCount);
    printf("  Total amount of constant memory:               %4.2f KB\n", iProp.totalConstMem / 1024.0);
    printf("  Total amount of shared memory per block:       %4.2f KB\n", iProp.sharedMemPerBlock / 1024.0);
    printf("  Total number of registers available per block: %d\n", iProp.regsPerBlock);
    printf("  Warp size:                                     %d\n", iProp.warpSize);
    printf("  Maximum number of threads per block:           %d\n", iProp.maxThreadsPerBlock);
    printf("  Maximum number of threads per multiprocessor:  %d\n", iProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of warps per multiprocessor:    %d\n", iProp.maxThreadsPerMultiProcessor / 32);
}
__global__ void mathKernel1(float* c)
{
    int   tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    if (tid % 2 == 0) {
        ia = 100.0f;
    }
    else {
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

__global__ void mathKernel2(float* c)
{
    int   tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    if ((tid / warpSize) % 2 == 0) {
        ia = 100.0f;
    }
    else {
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

__global__ void mathKernel3(float* c)
{
    int   tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib    = 0.0f;
    bool ipred = (tid % 2 == 0);

    if (ipred) {
        ia = 100.0f;
    }
    if (!ipred) {
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

__global__ void mathKernel4(float* c)
{
    int   tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib  = 0.0f;
    int itid = tid >> 5;
    if (itid & 0x01 == 0) {
        ia = 100.0f;
    }
    else {
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

__global__ void warmingup(float* c)
{
    int   tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    if ((tid / warpSize) % 2 == 0) {
        ia = 100.0f;
    }
    else {
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}



void simpleDivergence()
{

    int size      = 1 << 12;
    int blocksize = 128;
    // if(argc > 1) blocksize = atoi(argv[1]);
    // if(argc > 2) size      = atoi(argv[2]);
    printf("Data size %d ", size);

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    //
    float* d_C;
    size_t nBytes = size * sizeof(float);
    CHECK(hipMalloc((float**)&d_C, nBytes));
    // run a warmup kernel to remove overhead
    CHECK(hipDeviceSynchronize());

    TICK(warmup)
    warmingup<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    TOCK(warmup)

    // run kernel 1
    TICK(mathKernel1)
    mathKernel1<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    TOCK(mathKernel1)

    // run kernel 3
    TICK(mathKernel2)
    mathKernel2<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    TOCK(mathKernel2)


    // run kernel 3
    TICK(mathKernel3)
    mathKernel3<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    TOCK(mathKernel3)

    // run kernel 4
    TICK(mathKernel4)
    mathKernel4<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    TOCK(mathKernel4)

    CHECK(hipGetLastError());
    CHECK(hipFree(d_C));
    CHECK(hipDeviceReset());
}


__global__ void nestedHelloWorld(int const iSize, int iDepth)
{
    int tid = threadIdx.x;
    printf("Recursion=%d: Hello World from thread %d block %d\n", iDepth, tid, blockIdx.x);
    // condition to stop recursive execution
    if (iSize == 1)
        return;
    // reduce block size to half
    int nthreads = iSize >> 1;
    // thread 0 launches child grid recursively
    if (tid == 0 && nthreads > 0) {
        nestedHelloWorld<<<1, nthreads>>>(nthreads, ++iDepth);
        printf("-------> nested execution depth: %d\n", iDepth);
    }
}

void nestedHelloWorld()
{
    int size      = 8;
    int blocksize = 8;   // initial block size
    int igrid     = 1;
    // if(argc > 1)
    // {
    //     igrid = atoi(argv[1]);
    //     size = igrid * blocksize;
    // }
    // igrid = 4;
    // size = igrid * blocksize;
    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    printf("Execution Configuration: grid %d block %d\n", grid.x, block.x);
    nestedHelloWorld<<<grid, block>>>(block.x, 0);
    CHECK(hipGetLastError());
    CHECK(hipDeviceReset());
}

// Recursive Implementation of Interleaved Pair Approach
int cpuRecursiveReduce(int* data, int const size)
{
    // stop condition
    if (size == 1)
        return data[0];
    // renew the stride
    int const stride = size / 2;
    // in-place reduction
    for (int i = 0; i < stride; i++) {
        data[i] += data[i + stride];
    }
    // call recursively
    return cpuRecursiveReduce(data, stride);
}

// Neighbored Pair Implementation with divergence
__global__ void reduceNeighbored(int* g_idata, int* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x;
    // boundary check
    if (idx >= n)
        return;
    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            idata[tid] += idata[tid + stride];
        }
        // synchronize within threadblock
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void gpuRecursiveReduce(int* g_idata, int* g_odata, unsigned int isize)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x;
    int* odata = &g_odata[blockIdx.x];
    // stop condition
    if (isize == 2 && tid == 0) {
        g_odata[blockIdx.x] = idata[0] + idata[1];
        return;
    }
    // nested invocation
    int istride = isize >> 1;
    if (istride > 1 && tid < istride) {
        // in place reduction
        idata[tid] += idata[tid + istride];
    }
    // sync at block level
    __syncthreads();
    // nested invocation to generate child grids
    if (tid == 0) {
        gpuRecursiveReduce<<<1, istride>>>(idata, odata, istride);
        // sync all child grids launched in this block
        // hipDeviceSynchronize();
    }
    // sync at block level again
    __syncthreads();
}

__global__ void gpuRecursiveReduceNosync(int* g_idata, int* g_odata, unsigned int isize)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x;
    int* odata = &g_odata[blockIdx.x];
    // stop condition
    if (isize == 2 && tid == 0) {
        g_odata[blockIdx.x] = idata[0] + idata[1];
        return;
    }
    // nested invoke
    int istride = isize >> 1;
    if (istride > 1 && tid < istride) {
        idata[tid] += idata[tid + istride];
        if (tid == 0) {
            gpuRecursiveReduceNosync<<<1, istride>>>(idata, odata, istride);
        }
    }
}

__global__ void gpuRecursiveReduce2(int* g_idata, int* g_odata, int iStride, int const iDim)
{
    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * iDim;
    // stop condition
    if (iStride == 1 && threadIdx.x == 0) {
        g_odata[blockIdx.x] = idata[0] + idata[1];
        return;
    }
    // in place reduction
    idata[threadIdx.x] += idata[threadIdx.x + iStride];
    // nested invocation to generate child grids
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        gpuRecursiveReduce2<<<gridDim.x, iStride / 2>>>(g_idata, g_odata, iStride / 2, iDim);
    }
}

void nestedReduce2()
{

    // set up device
    // int dev = 0;
    int gpu_sum;
    // hipDeviceProp_t deviceProp;
    // CHECK(hipGetDeviceProperties(&deviceProp, dev));
    // printf("%s starting reduction at ", argv[0]);
    // printf("device %d: %s ", dev, deviceProp.name);
    // CHECK(hipSetDevice(dev));

    bool bResult = false;
    // set up execution configuration
    int nblock  = 2048;
    int nthread = 512;   // initial block size

    // if(argc > 1){
    //     nblock = atoi(argv[1]);   // block size from command line argument
    // }

    // if(argc > 2){
    //     nthread = atoi(argv[2]);   // block size from command line argument
    // }

    int size = nblock * nthread;   // total number of elements to reduceNeighbored

    dim3 block(nthread, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    printf("array %d grid %d block %d\n", size, grid.x, block.x);

    // allocate host memory
    size_t bytes   = size * sizeof(int);
    int*   h_idata = (int*)malloc(bytes);
    int*   h_odata = (int*)malloc(grid.x * sizeof(int));
    int*   tmp     = (int*)malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++) {
        h_idata[i] = (int)(rand() & 0xFF);
        h_idata[i] = 1;
    }

    memcpy(tmp, h_idata, bytes);

    // allocate device memory
    int* d_idata = NULL;
    int* d_odata = NULL;
    CHECK(hipMalloc((void**)&d_idata, bytes));
    CHECK(hipMalloc((void**)&d_odata, grid.x * sizeof(int)));



    // cpu recursive reduction
    TICK(cpu_reduce)
    int cpu_sum = cpuRecursiveReduce(tmp, size);
    TOCK(cpu_reduce)
    printf("cpu reduce\t\telapsed cpu_sum: %d\n", cpu_sum);

    // gpu reduceNeighbored
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    TICK(gpu_Neighbored)
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    TOCK(gpu_Neighbored)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_odata[i];
    printf("gpu Neighbored\t\telapsed gpu_sum: %d <<<grid %d block %d>>>\n", gpu_sum, grid.x, block.x);


    // gpu nested reduce kernel
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    TICK(gpu_nested)
    gpuRecursiveReduce<<<grid, block>>>(d_idata, d_odata, block.x);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    TOCK(gpu_nested)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_odata[i];
    printf("gpu nested\t\telapsed gpu_sum: %d <<<grid %d block %d>>>\n", gpu_sum, grid.x, block.x);

    // gpu nested reduce kernel without synchronization
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    TICK(gpu_nested_without_synchronization)
    gpuRecursiveReduceNosync<<<grid, block>>>(d_idata, d_odata, block.x);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    TOCK(gpu_nested_without_synchronization)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_odata[i];
    printf("gpu nestedNosyn\t\telapsed gpu_sum: %d <<<grid %d block %d>>>\n", gpu_sum, grid.x, block.x);


    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    TICK(gpu_nested2)
    gpuRecursiveReduce2<<<grid, block.x / 2>>>(d_idata, d_odata, block.x / 2, block.x);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    TOCK(gpu_nested2)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_odata[i];
    printf("gpu nested2\t\telapsed gpu_sum: %d <<<grid %d block %d>>>\n", gpu_sum, grid.x, block.x);

    // free host memory
    free(h_idata);
    free(h_odata);
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));
    CHECK(hipDeviceReset());
    bResult = (gpu_sum == cpu_sum);
    if (!bResult)
        printf("Test failed!\n");
}

// Recursive Implementation of Interleaved Pair Approach
int recursiveReduce(int* data, int const size)
{
    // terminate check
    if (size == 1)
        return data[0];
    // renew the stride
    int const stride = size / 2;
    // in-place reduction
    for (int i = 0; i < stride; i++) {
        data[i] += data[i + stride];
    }
    // call recursively
    return recursiveReduce(data, stride);
}

// Neighbored Pair Implementation with less divergence
__global__ void reduceNeighboredLess(int* g_idata, int* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n)
        return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        // convert tid into local array index
        int index = 2 * stride * tid;

        if (index < blockDim.x) {
            idata[index] += idata[index + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

// Interleaved Pair Implementation with less divergence
__global__ void reduceInterleaved(int* g_idata, int* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n)
        return;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrolling2(int* g_idata, int* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 2;

    // unrolling 2
    if (idx + blockDim.x < n)
        g_idata[idx] += g_idata[idx + blockDim.x];

    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrolling4(int* g_idata, int* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 4;

    // unrolling 4
    if (idx + 3 * blockDim.x < n) {
        int a1       = g_idata[idx];
        int a2       = g_idata[idx + blockDim.x];
        int a3       = g_idata[idx + 2 * blockDim.x];
        int a4       = g_idata[idx + 3 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4;
    }

    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrolling8(int* g_idata, int* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        int a1       = g_idata[idx];
        int a2       = g_idata[idx + blockDim.x];
        int a3       = g_idata[idx + 2 * blockDim.x];
        int a4       = g_idata[idx + 3 * blockDim.x];
        int b1       = g_idata[idx + 4 * blockDim.x];
        int b2       = g_idata[idx + 5 * blockDim.x];
        int b3       = g_idata[idx + 6 * blockDim.x];
        int b4       = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrollWarps8(int* g_idata, int* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        int a1       = g_idata[idx];
        int a2       = g_idata[idx + blockDim.x];
        int a3       = g_idata[idx + 2 * blockDim.x];
        int a4       = g_idata[idx + 3 * blockDim.x];
        int b1       = g_idata[idx + 4 * blockDim.x];
        int b2       = g_idata[idx + 5 * blockDim.x];
        int b3       = g_idata[idx + 6 * blockDim.x];
        int b4       = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32) {
        volatile int* vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceCompleteUnrollWarps8(int* g_idata, int* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        int a1       = g_idata[idx];
        int a2       = g_idata[idx + blockDim.x];
        int a3       = g_idata[idx + 2 * blockDim.x];
        int a4       = g_idata[idx + 3 * blockDim.x];
        int b1       = g_idata[idx + 4 * blockDim.x];
        int b2       = g_idata[idx + 5 * blockDim.x];
        int b3       = g_idata[idx + 6 * blockDim.x];
        int b4       = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512)
        idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)
        idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128)
        idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)
        idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int* vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

template<unsigned int iBlockSize>
__global__ void reduceCompleteUnroll(int* g_idata, int* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        int a1       = g_idata[idx];
        int a2       = g_idata[idx + blockDim.x];
        int a3       = g_idata[idx + 2 * blockDim.x];
        int a4       = g_idata[idx + 3 * blockDim.x];
        int b1       = g_idata[idx + 4 * blockDim.x];
        int b2       = g_idata[idx + 5 * blockDim.x];
        int b3       = g_idata[idx + 6 * blockDim.x];
        int b4       = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (iBlockSize >= 1024 && tid < 512)
        idata[tid] += idata[tid + 512];

    __syncthreads();

    if (iBlockSize >= 512 && tid < 256)
        idata[tid] += idata[tid + 256];

    __syncthreads();

    if (iBlockSize >= 256 && tid < 128)
        idata[tid] += idata[tid + 128];

    __syncthreads();

    if (iBlockSize >= 128 && tid < 64)
        idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int* vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrollWarps(int* g_idata, int* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x * 2;

    // unrolling 2
    if (idx + blockDim.x < n)
        g_idata[idx] += g_idata[idx + blockDim.x];

    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // unrolling last warp
    if (tid < 32) {
        volatile int* vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}

void reduceInteger( int argv1)
{
    // set up device
    // int            dev = 0;
    // hipDeviceProp_t deviceProp;
    // CHECK(hipGetDeviceProperties(&deviceProp, dev));
    // printf("%s starting reduction at ", argv[0]);
    // printf("device %d: %s ", dev, deviceProp.name);
    // CHECK(hipSetDevice(dev));

    bool bResult = false;

    // initialization
    int size = 1 << 24;   // total number of elements to reduce
    printf("    with array size %d  ", size);

    // execution configuration
    int blocksize = 512;   // initial block size

    if (argv1 > 1) {
        blocksize = argv1;   // block size from command line argument
    }

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes   = size * sizeof(int);
    int*   h_idata = (int*)malloc(bytes);
    int*   h_odata = (int*)malloc(grid.x * sizeof(int));
    int*   tmp     = (int*)malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++) {
        // mask off high 2 bytes to force max number to 255
        h_idata[i] = (int)(rand() & 0xFF);
    }

    memcpy(tmp, h_idata, bytes);
    int    gpu_sum = 0;

    // allocate device memory
    int* d_idata = NULL;
    int* d_odata = NULL;
    CHECK(hipMalloc((void**)&d_idata, bytes));
    CHECK(hipMalloc((void**)&d_odata, grid.x * sizeof(int)));

    // cpu reduction
    TICK(recursiveReduce)
    int cpu_sum = recursiveReduce(tmp, size);
    TOCK(recursiveReduce)
    printf("cpu reduce cpu_sum: %d\n", cpu_sum);

    // kernel 1: reduceNeighbored
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    TICK(reduceNeighbored)
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    TOCK(reduceNeighbored)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_odata[i];

    printf("gpu Neighbored  gpu_sum: %d <<<grid %d block %d>>>\n",gpu_sum,grid.x,block.x);


    // kernel 2: reduceNeighbored with less divergence
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    TICK(reduceNeighboredLess)
    reduceNeighboredLess<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    TOCK(reduceNeighboredLess)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_odata[i];

    printf("gpu Neighbored2 gpu_sum: %d <<<grid %d block %d>>>\n",gpu_sum,grid.x,block.x);

    // kernel 3: reduceInterleaved
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    TICK(reduceInterleaved)
    reduceInterleaved<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    TOCK(reduceInterleaved)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++)
        gpu_sum += h_odata[i];

    printf("gpu Interleaved gpu_sum: %d <<<grid %d block %d>>>\n",gpu_sum,grid.x,block.x);

    // kernel 4: reduceUnrolling2
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    TICK(reduceUnrolling2)
    reduceUnrolling2<<<grid.x / 2, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    TOCK(reduceUnrolling2)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 2 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 2; i++)
        gpu_sum += h_odata[i];

    printf("gpu Unrolling2  gpu_sum: %d <<<grid %d block %d>>>\n",gpu_sum,grid.x / 2,block.x);

    // kernel 5: reduceUnrolling4
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    TICK(reduceUnrolling4)
    reduceUnrolling4<<<grid.x / 4, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    TOCK(reduceUnrolling4)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 4 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 4; i++)
        gpu_sum += h_odata[i];

    printf("gpu Unrolling4  gpu_sum: %d <<<grid %d block %d>>>\n",gpu_sum,grid.x / 4,block.x);

    // kernel 6: reduceUnrolling8
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    TICK(reduceUnrolling8)
    reduceUnrolling8<<<grid.x / 8, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    TOCK(reduceUnrolling8)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += h_odata[i];

    printf("gpu Unrolling8  gpu_sum: %d <<<grid %d block %d>>>\n",gpu_sum,grid.x / 8,block.x);

    for (int i = 0; i < grid.x / 16; i++)
        gpu_sum += h_odata[i];

    // kernel 8: reduceUnrollWarps8
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    TICK(reduceUnrollWarps8)
    reduceUnrollWarps8<<<grid.x / 8, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    TOCK(reduceUnrollWarps8)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += h_odata[i];

    printf("gpu UnrollWarp8 gpu_sum: %d <<<grid %d block %d>>>\n",gpu_sum,grid.x / 8,block.x);


    // kernel 9: reduceCompleteUnrollWarsp8
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    TICK(reduceCompleteUnrollWarps8)
    reduceCompleteUnrollWarps8<<<grid.x / 8, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    TOCK(reduceCompleteUnrollWarps8)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += h_odata[i];

    printf("gpu Cmptnroll8  gpu_sum: %d <<<grid %d block %d>>>\n",gpu_sum,grid.x / 8,block.x);

    // kernel 9: reduceCompleteUnroll
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());

    TICK(reduceCompleteUnroll)
    switch (blocksize) {
    case 1024:
        reduceCompleteUnroll<1024><<<grid.x / 8, block>>>(d_idata, d_odata, size);
        break;

    case 512:
        reduceCompleteUnroll<512><<<grid.x / 8, block>>>(d_idata, d_odata, size);
        break;

    case 256:
        reduceCompleteUnroll<256><<<grid.x / 8, block>>>(d_idata, d_odata, size);
        break;

    case 128:
        reduceCompleteUnroll<128><<<grid.x / 8, block>>>(d_idata, d_odata, size);
        break;

    case 64:
        reduceCompleteUnroll<64><<<grid.x / 8, block>>>(d_idata, d_odata, size);
        break;
    }

    CHECK(hipDeviceSynchronize());
    TOCK(reduceCompleteUnroll)
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));

    gpu_sum = 0;

    for (int i = 0; i < grid.x / 8; i++)
        gpu_sum += h_odata[i];

    printf("gpu Cmptnroll   gpu_sum: %d <<<grid %d block %d>>>\n",gpu_sum,grid.x / 8,block.x);

    // free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    // reset device
    CHECK(hipDeviceReset());

    // check the results
    bResult = (gpu_sum == cpu_sum);

    if (!bResult)
        printf("Test failed!\n");
}