#include "hip/hip_runtime.h"
﻿#include "test_03.cuh"

void simpleDeviceQuery(){
    int iDev = 0;
    hipDeviceProp_t iProp;
    CHECK(hipGetDeviceProperties(&iProp, iDev));
    printf("Device %d: %s\n", iDev, iProp.name);
    printf("  Number of multiprocessors:                     %d\n",iProp.multiProcessorCount);
    printf("  Total amount of constant memory:               %4.2f KB\n",iProp.totalConstMem / 1024.0);
    printf("  Total amount of shared memory per block:       %4.2f KB\n",iProp.sharedMemPerBlock / 1024.0);
    printf("  Total number of registers available per block: %d\n",iProp.regsPerBlock);
    printf("  Warp size:                                     %d\n",iProp.warpSize);
    printf("  Maximum number of threads per block:           %d\n",iProp.maxThreadsPerBlock);
    printf("  Maximum number of threads per multiprocessor:  %d\n",iProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of warps per multiprocessor:    %d\n",iProp.maxThreadsPerMultiProcessor / 32);


}
__global__ void mathKernel1(float *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    if (tid % 2 == 0){
        ia = 100.0f;
    }
    else{
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

__global__ void mathKernel2(float *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    if ((tid / warpSize) % 2 == 0){
        ia = 100.0f;
    }
    else{
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

__global__ void mathKernel3(float *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    bool ipred = (tid % 2 == 0);

    if (ipred){
        ia = 100.0f;
    }
    if (!ipred){
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

__global__ void mathKernel4(float *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    int itid = tid >> 5;
    if (itid & 0x01 == 0){
        ia = 100.0f;
    }
    else{
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

__global__ void warmingup(float *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    if ((tid / warpSize) % 2 == 0){
        ia = 100.0f;
    }
    else {
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}



void simpleDivergence(){

    int size = 1<< 12;
    int blocksize = 128;
    // if(argc > 1) blocksize = atoi(argv[1]);
    // if(argc > 2) size      = atoi(argv[2]);
    printf("Data size %d ", size);

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    //
    float *d_C;
    size_t nBytes = size * sizeof(float);
    CHECK(hipMalloc((float**)&d_C, nBytes));
    // run a warmup kernel to remove overhead
    CHECK(hipDeviceSynchronize());

    TICK(warmup)
    warmingup<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    TOCK(warmup)

    // run kernel 1
    TICK(mathKernel1)
    mathKernel1<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    TOCK(mathKernel1)

    // run kernel 3
    TICK(mathKernel2)
    mathKernel2<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    TOCK(mathKernel2)


    // run kernel 3
    TICK(mathKernel3)
    mathKernel3<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    TOCK(mathKernel3)

    // run kernel 4
    TICK(mathKernel4)
    mathKernel4<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    TOCK(mathKernel4)

    CHECK(hipGetLastError());
    CHECK(hipFree(d_C));
    CHECK(hipDeviceReset());
}