﻿#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition without privatization
__global__ void vectorAdd(const float *a, const float *b, float *result, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        result[index] = a[index] + b[index];
    }
}

// CUDA kernel for vector addition with privatization
__global__ void vectorAddPrivatized(const float *a, const float *b, float *result, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        float a_private = a[index]; // Load into private memory
        float b_private = b[index]; // Load into private memory
        result[index] = a_private + b_private;
    }
}

// Function to initialize the vectors with dummy data
void initData(float *data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = i;
    }
}

int main() {
    int n = 1<<24; // Size of the vectors
    float *a, *b, *result, *d_a, *d_b, *d_result;

    // Allocate memory on the host
    a = (float*)malloc(n * sizeof(float));
    b = (float*)malloc(n * sizeof(float));
    result = (float*)malloc(n * sizeof(float));

    // Initialize vectors
    initData(a, n);
    initData(b, n);

    // Allocate memory on the device
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_result, n * sizeof(float));

    hipEvent_t start, stop, startCoarsened, stopCoarsened;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startCoarsened);
    hipEventCreate(&stopCoarsened);


    // Copy vectors from host to device
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Define number of blocks and threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the vector addition kernel without privatization
    hipEventRecord(start);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("vectorAdd execution time: %f ms\n", milliseconds);

    // Copy result back to host
    hipMemcpy(result, d_result, n * sizeof(float), hipMemcpyDeviceToHost);

    // Launch the vector addition kernel with privatization
    hipEventRecord(startCoarsened);
    vectorAddPrivatized<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, n);
    hipEventRecord(stopCoarsened);
    hipEventSynchronize(stopCoarsened);
    float millisecondsCoarsened = 0;
    hipEventElapsedTime(&millisecondsCoarsened, startCoarsened, stopCoarsened);
    printf("vectorAddPrivatized execution time: %f ms\n", millisecondsCoarsened);

    // Copy result back to host
    hipMemcpy(result, d_result, n * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
    free(a);
    free(b);
    free(result);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(startCoarsened);
    hipEventDestroy(stopCoarsened);

    return 0;
}
